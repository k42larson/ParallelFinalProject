#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//number of total simulations
#define N   200000

__global__ void add( int *GoingWon, int *NotGoingWon, unsigned int seed ) {
	//get the id of the block
	int tid = blockIdx.x;

	//initalize random number generator
	hiprandState_t state;
	hiprand_init(seed, tid, 0, &state);

	//the initial score of the game
	//team 1 is user's team, team 2 is the opponent
	int team1Score = 17;
	int team2Score = 10;

	//current position of the ball on the field
	//in terms of yards from your endzone
	double position = 50;

	//current down(always starts at 4)
	int down = 4;

	//yardToGet to get first down
	//in terms of yards from your endzone
	double yardToGet = 55;

	//current quarter and seconds left in the quarter
	//(5 for ot)
	int quarter = 4;
	int secondsLeft = 400;

	//Did user's team get the ball first?
	bool team1BallFirst = true;
	
	
	//offense and defense information
	//league average is approximately 5.6
	double averageYardsTeam1 = 5.6;
	double averageYardsTeam2 = 5.6;
	double averageYardsGivenTeam1 = 5.6;
	double averageYardsGivenTeam2 = 5.6;

	//variable to keep track of team in possesion of the ball
	bool team1HasBall = true;

	//variable to keep track of the game ending	
	bool gameOver = false;
	
	//variable to keep track of the simulated play length
	double playLength = 0;
	
	//variable to figure out whether this simulation
	//will be going for it or not
	int whichScenario = tid % 2;
	
	//variable to keep track if this is the first play
	//of the simulation
	bool isFirstPlay = true;

	//the simulated game loop
	while(!gameOver)
	{
		//since the simulation defaults to not going
		//for it on 4th down, we must override on the
		//first play
		if(down < 4 || (isFirstPlay && whichScenario == 0))
		{
			//this if statement generates the amount of yards gained 
			//on the play using a random number between 0 and 1
			double x = hiprand_uniform(&state);
			if(x <= .2)
			{
				playLength = 22.36 * sqrt(x) - 10;
			}
			else if(x <= .8)
			{
				playLength = 14.3 * x - 2.86;
			}
			else
			{
				playLength =  300 * (x - .7) * (x - .7) + 5.86;
			}
			
			//determine who has the ball before determining the effects of the play
			//team 1 has ball
			if(team1HasBall)
			{
				
				//adjust based on the team 1's offense and team 2's defense
				playLength += averageYardsTeam1 + averageYardsGivenTeam2 - 11.2;
				
				//update position
				position += playLength;
				//if touchdown
				if(position >= 100)
			        {
					team1Score += 7;
					team1HasBall = false;
					//for simplicity's sake kickoffs will always be touchbacks
					position = 75;
					yardToGet = 65;
					down = 1;
					//this if statement is to check for ot
					//and end it since any score ends ot
					if(quarter > 4)
						gameOver = true;
			        }
				//failed to get first down
				else if(position < yardToGet)
					down++;
				//saftey
				else if(position <= 0)
				{
					team2Score += 2;
					team1HasBall = false;
					position = 75;
					yardToGet = 65;
					down = 1;
					if(quarter > 4)
						gameOver = true;
				}
				//first down
				else
				{
					yardToGet = position + 10;
					//check for first and goal
					if(yardToGet > 100)
					{
						yardToGet = 100;
					}
					down = 1;
				}
			}
			else
			//team 2 has ball
			{
				//adjust based on the team 2's offense and team 1's defense
				playLength += averageYardsTeam2 + averageYardsGivenTeam1 - 11.2;
				//update position
				position -= playLength;

				//if touchdown
				if(position <= 0)
				{
					team2Score += 7;
					team1HasBall = true;
					position = 25;
					yardToGet = 35;
					down = 1;
					if(quarter > 4)
						gameOver = true;
				}
				//safety
				else if(position >= 100)
				{
					team1Score += 2;
					team1HasBall = true;
					position = 25;
					yardToGet = 35;
					down = 1;
					if(quarter > 4)
						gameOver = true;
				}
				//failed to get first down
				else if(position > yardToGet)
					down++;
				//first down
				else
				{
					yardToGet = position - 10;
					//first and goal
					if(yardToGet < 0)
					{
						yardToGet = 0;
					}
					down = 1;
				}
			}
		}
		//failed 4th down conversion
		else if(down >= 5)
		{
			if(team1HasBall)
			{
				yardToGet = position - 10;
				down = 1;
				team1HasBall = false;
			}
			else
			{
				yardToGet = position + 10;
				down = 1;
				team1HasBall = true;
			}
		}
		//it is typical a 4th down
		else
		{
			//team 1 has ball
			if(team1HasBall)
			{
				//out of field goal range
				if(position < 65)
				{
					//the average net punt is roughly 40 yards
					position += 40;
					//check for touchback
					if(position >= 100)
					{
						position = 75;
					}
				}
				else
				{
					//generate random number to see if field goal
					double y = hiprand_uniform(&state);
					//odds of making a field goal roughly correlates with
					// 1 percentage point per yard away from the end zone
					if(y < position/100.0)
					{
						team1Score += 3;
						position = 75;
						if(quarter > 4)
							gameOver = true;
					}
					//if the field goal is missed the position doesn't 
					//change unless the team is inside the 25
					//then it acts like a touchback
					else if(position > 75)
					{
						position = 75;
					}
				}
				//update the first down line, down and possesion
				yardToGet = position - 10;
				down = 1;
				team1HasBall = false;
			}
			//team 2 has ball
			else
			{
				//out of field goal range
				if(position > 35)
				{	
					//the average net punt is roughly 40 yards
					position -= 40;
					//check for touchback
					if(position <= 0)
					{
						position = 25;
					}
				}
				else
				{
					//generate random number to see if field goal
					double y = hiprand_uniform(&state);
					//odds of making a field goal roughly correlates with
					// 1 percentage point per yard away from the end zone
					if(y < (100-position)/100.0)
					{
						team2Score += 3;
						position = 25;
						if(quarter > 4)
							gameOver = true;
					}
					//if the field goal is missed the position doesn't 
					//change unless the team is inside the 25
					//then it acts like a touchback
					else if(position < 25)
					{
						position = 25;
					}
				}
				//update the first down line, down and possesion
				yardToGet = position + 10;
				down = 1;
				team1HasBall = true;
			}
		}
		//each play takes about 25 seconds
		//because there are roughly 150 plays per game 
		secondsLeft -= 20;
		//end of the quarter 
		if(secondsLeft <= 0)
		{
			quarter++;
			//check to see if the game ended
			if(quarter >= 5 && team1Score != team2Score)
			{
				gameOver = true;
			}
			//end of the half
			if(quarter == 3)
			{
				//figure out who which team got the ball first
				if(team1BallFirst)
				{
					position = 75;
					yardToGet = 65;
					down = 1;
					team1HasBall = false;
				}
				else
				{
					position = 25;
					yardToGet = 35;
					down = 1;
					team1HasBall = true;
				}
				secondsLeft = 900;
			}
			//reset the quarter countdown
			else
			{
				secondsLeft = 900;
			}
		}
		//no longer first play
		isFirstPlay = false;
		/*if(tid == 68)
		{
			printf("%d %d\n", team1Score, team2Score);
			printf("%f %d %f\n", position, down, yardToGet);
			printf("%d %d\n\n", quarter, secondsLeft);
		}*/
	}
	if(team1Score > team2Score)
	{
		if(whichScenario == 0)
			GoingWon[tid] = 1;
		else
			NotGoingWon[tid] = 1;
	}
}

int main( void ) {
    	int GoingWon[N], NotGoingWon[N];
    	int *dev_a, *dev_b ;
	

    	// allocate the memory on the GPU
    	hipMalloc( (void**)&dev_a, N * sizeof(int) );
    	hipMalloc( (void**)&dev_b, N * sizeof(int) );


    	// fill the arrays 'a' and 'b' on the CPU
    	for (int i=0; i<N; i++) {
        	GoingWon[i] = 0;
        	NotGoingWon[i] = 0;
   	}

    	// copy the arrays 'a' and 'b' to the GPU
    	hipMemcpy( dev_a, GoingWon, N * sizeof(int),
                              hipMemcpyHostToDevice );
    	hipMemcpy( dev_b, NotGoingWon, N * sizeof(int),
                              hipMemcpyHostToDevice );

    	add<<<N,1>>>( dev_a, dev_b, time(NULL) );

    	// copy the array 'c' back from the GPU to the CPU
    	hipMemcpy( GoingWon, dev_a, N * sizeof(int),
                              hipMemcpyDeviceToHost );

	hipMemcpy( NotGoingWon, dev_b, N * sizeof(int),
                              hipMemcpyDeviceToHost );

    	// calculate the results
	int totGoingWon = 0;
	int totNotGoingWon = 0;
    	for (int i=0; i<N; i++) {
        	totGoingWon += GoingWon[i];
		totNotGoingWon += NotGoingWon[i];
    	}
	printf("How often you won going for it: %f%%\nHow often you won not going for it: %f%%\n", 200.0*totGoingWon/N, 200.0*totNotGoingWon/N);

    	// free the memory allocated on the GPU
    	hipFree( dev_a ) ;
    	hipFree( dev_b ) ;

    	return 0;
}